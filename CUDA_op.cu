#include "hip/hip_runtime.h"
// CUDA compilation fix
#ifdef __HIPCC__
#define _USE_MATH_DEFINES
#include <math.h>
#endif

#include "CUDA_op.cuh"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include "Global.h"

// Forward declarations to avoid including AcqTrk.h
void f_Peak_Detector(Channel_struct& f_CH, double* Z_IFFT);
void f_Peak_Detector_5ms(Channel_struct& f_CH, double* Z_IFFT);

// #include "AcqTrk.h"  // Commented out to avoid stdafx.h dependency


#define CorrSizeLocal SAMPLING_FREQ / 1000.0		//�⺻�� : 50000
#define Trk_CorrSizeLocal (SAMPLING_FREQ / 1000.0) + 10		//�⺻�� : 50000
#define BATCH 1
#define SAMPLING_FREQ	Sampling_Frequency
#define DIM_BLOCK_COL	512
#define F_CA_GPS		1023000
#define IF_FREQ			IF_Frequency

__constant__ int Gpu_const_f_s[1] = { 0, };
__constant__ int Gpu_const_f_if_GPS[1] = { 0, };
__constant__ int Gpu_const_f_ca_GPS[1] = { 0, };
__constant__ int Gpu_const_sample_size_1ms[1] = { 0, };


__global__ void f_Replica_Gen(hipfftDoubleComplex* replica_carr_vec, hipfftDoubleComplex* replica_code_vec, short* IF_Data, short* Range_dopp, int* CA_code_GPS_1023, short* PRN)
{
	int thread_num = blockDim.x * blockIdx.x + threadIdx.x;
	int CH_PRN = blockIdx.y / 45;
	short ifdata_shift = IF_Data[thread_num] >> 8;

	if (thread_num > (int)(SAMPLING_FREQ / 1000.0))	return;

	int dopp_f = Range_dopp[blockIdx.y % 45];

	int temp = ((int)((double)thread_num / (double)SAMPLING_FREQ * (double)F_CA_GPS)) % 1023;
	int element_num = thread_num + ((int)(SAMPLING_FREQ / 1000.0) * blockIdx.y);

	replica_carr_vec[element_num].x = ifdata_shift * cos(2.0 * M_PI * (IF_FREQ + dopp_f) * thread_num / SAMPLING_FREQ);
	replica_carr_vec[element_num].y = ifdata_shift * sin(2.0 * M_PI * (IF_FREQ + dopp_f) * thread_num / SAMPLING_FREQ);
	replica_code_vec[element_num].x = CA_code_GPS_1023[(PRN[CH_PRN] - 1) * 1023 + temp];
	replica_code_vec[element_num].y = 0.0;
}

__global__ void f_Replica_Gen_5ms(hipfftDoubleComplex* replica_carr_vec, hipfftDoubleComplex* replica_code_vec, short* IF_Data, short* Range_dopp, int* CA_code_GPS_1023, short* PRN)
{
	int thread_num = blockDim.x * blockIdx.x + threadIdx.x;
	int CH_PRN = blockIdx.y / 45;
	
	if (thread_num >= (int)(SAMPLING_FREQ * 5 / 1000.0))	return;  // 5ms boundary check
	
	short ifdata_shift = IF_Data[thread_num] >> 8;
	int dopp_f = Range_dopp[blockIdx.y % 45];

	// For 5ms, we need to repeat the CA code 5 times (5 x 1023 chips)
	int temp = ((int)((double)thread_num / (double)SAMPLING_FREQ * (double)F_CA_GPS)) % 1023;
	int element_num = thread_num + ((int)(SAMPLING_FREQ * 5 / 1000.0) * blockIdx.y);

	replica_carr_vec[element_num].x = ifdata_shift * cos(2.0 * M_PI * (IF_FREQ + dopp_f) * thread_num / SAMPLING_FREQ);
	replica_carr_vec[element_num].y = ifdata_shift * sin(2.0 * M_PI * (IF_FREQ + dopp_f) * thread_num / SAMPLING_FREQ);
	replica_code_vec[element_num].x = CA_code_GPS_1023[(PRN[CH_PRN] - 1) * 1023 + temp];
	replica_code_vec[element_num].y = 0.0;
}

__global__ void f_Freq_Conjugate(hipfftDoubleComplex* freq_Mul_conj_carr_code, hipfftDoubleComplex* freq_replica_carr_vec, hipfftDoubleComplex* freq_replica_code_vec)
{
	int thread_num = blockDim.x * blockIdx.x + threadIdx.x;
	if (thread_num > (int)(SAMPLING_FREQ / 1000.0))	return;

	int element_num = thread_num + ((int)(SAMPLING_FREQ / 1000.0) * blockIdx.y);

	freq_Mul_conj_carr_code[element_num].x = (freq_replica_code_vec[element_num].x * freq_replica_carr_vec[element_num].x) + (freq_replica_code_vec[element_num].y * freq_replica_carr_vec[element_num].y);
	freq_Mul_conj_carr_code[element_num].y = (freq_replica_code_vec[element_num].x * freq_replica_carr_vec[element_num].y) - (freq_replica_code_vec[element_num].y * freq_replica_carr_vec[element_num].x);


}

__global__ void f_Freq_Conjugate_5ms(hipfftDoubleComplex* freq_Mul_conj_carr_code, hipfftDoubleComplex* freq_replica_carr_vec, hipfftDoubleComplex* freq_replica_code_vec)
{
	int thread_num = blockDim.x * blockIdx.x + threadIdx.x;
	if (thread_num >= (int)(SAMPLING_FREQ * 5 / 1000.0))	return;  // 5ms boundary check

	int element_num = thread_num + ((int)(SAMPLING_FREQ * 5 / 1000.0) * blockIdx.y);

	freq_Mul_conj_carr_code[element_num].x = (freq_replica_code_vec[element_num].x * freq_replica_carr_vec[element_num].x) + (freq_replica_code_vec[element_num].y * freq_replica_carr_vec[element_num].y);
	freq_Mul_conj_carr_code[element_num].y = (freq_replica_code_vec[element_num].x * freq_replica_carr_vec[element_num].y) - (freq_replica_code_vec[element_num].y * freq_replica_carr_vec[element_num].x);
}


__global__ void f_Kernel_Abs(hipfftDoubleComplex* dump_IFFT, double* Z_IFFT)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int bch = blockIdx.y;	//doppler frequency
	int index = (bch * CorrSizeLocal) + tid;

	while (tid < (int)(*Gpu_const_sample_size_1ms))
	{
		Z_IFFT[index] = (dump_IFFT[index].x * dump_IFFT[index].x) + (dump_IFFT[index].y * dump_IFFT[index].y);

		tid += blockDim.x * gridDim.x;
	}
	__syncthreads();

}

__global__ void f_Kernel_Abs_5ms(hipfftDoubleComplex* dump_IFFT, double* Z_IFFT)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int bch = blockIdx.y;	//doppler frequency
	int sample_size_5ms = (int)(SAMPLING_FREQ * 5 / 1000.0);
	int index = (bch * sample_size_5ms) + tid;

	while (tid < sample_size_5ms)
	{
		Z_IFFT[index] = (dump_IFFT[index].x * dump_IFFT[index].x) + (dump_IFFT[index].y * dump_IFFT[index].y);

		tid += blockDim.x * gridDim.x;
	}
	__syncthreads();
}

void f_Acquisition(
	short* PRN,						// ���� PRN
	short* ifdata,					// if ������
	int* CA_code_GPS_1023,
	short* dopp_freq,
	hipfftDoubleComplex* replica_carr_vec,
	hipfftDoubleComplex* replica_code_vec,
	hipfftDoubleComplex* fft_out_carr_vec,
	hipfftDoubleComplex* fft_out_code_vec,
	hipfftDoubleComplex* freq_conj_carr_code,
	hipfftDoubleComplex* dump_IFFT,
	double* Z_IFFT,
	double* Cpu_Z_IFFT,
	int* ACQ_IF_DUMP,
	int* ACQ_MODE_V2,
	int* ACTIVECHANNEL,
	int* Cpu_code_phase
)
{

	int c = 0;

	dim3 grids(blocksPerGrid, doppler_bin_size * 32, 1);

	hipfftHandle p_cufft1, p_cufft2, p_cuifft;

	f_Replica_Gen << <grids, threadsPerBlock >> > (replica_carr_vec, replica_code_vec, ifdata, dopp_freq, CA_code_GPS_1023, PRN);

	int sampling_freq = SAMPLING_FREQ;
	int n_doppler_bin = doppler_bin_size;
	int batch = doppler_bin_size * 32;


	//FFT Plan
	hipfftPlan1d(&p_cufft1, (int)(SAMPLING_FREQ / 1000.0), HIPFFT_Z2Z, batch);
	hipfftPlan1d(&p_cufft2, (int)(SAMPLING_FREQ / 1000.0), HIPFFT_Z2Z, batch);
	hipfftPlan1d(&p_cuifft, (int)(SAMPLING_FREQ / 1000.0), HIPFFT_Z2Z, batch);


	//FFT ����
	hipfftExecZ2Z(p_cufft1, replica_carr_vec, fft_out_carr_vec, HIPFFT_FORWARD);
	hipfftExecZ2Z(p_cufft2, replica_code_vec, fft_out_code_vec, HIPFFT_FORWARD);

	hipDeviceSynchronize();

	f_Freq_Conjugate << <grids, threadsPerBlock >> > (freq_conj_carr_code, fft_out_carr_vec, fft_out_code_vec);

	//IFFT Plan
	hipfftExecZ2Z(p_cuifft, freq_conj_carr_code, dump_IFFT, HIPFFT_BACKWARD);

	hipDeviceSynchronize();


	f_Kernel_Abs << <grids, threadsPerBlock >> > (dump_IFFT, Z_IFFT);


	hipMemcpy(Cpu_Z_IFFT, Z_IFFT, sizeof(double) * CorrSizeLocal * doppler_bin_size * 32, hipMemcpyDeviceToHost);

	while ((*ACQ_IF_DUMP) == 1 && (*ACQ_MODE_V2) == 1)
	{

		f_Peak_Detector(CH[c], Cpu_Z_IFFT);

		if (CH[c].detect == true)
		{
			(*ACTIVECHANNEL)++;
			Cpu_code_phase[c] = CH[c].codephase;

			if (start_mode == false)
			{
				if (CH[c].PRN < 32)
				{
					CH[c + 1].PRN = CH[c].PRN + 1;
					c++;

				}
				else
				{
					(*ACQ_MODE_V2) = 0;
				}
			}
			else
			{
				c++;
			}
		}
		if (CH[c].ACQ_Check == false)
		{
			(*ACQ_MODE_V2) = 0;

		}
	}


	hipfftDestroy(p_cufft1);
	hipfftDestroy(p_cufft2);
	hipfftDestroy(p_cuifft);


	//return true;
}

void f_Acquisition_5ms(
	short* PRN,						// ���� PRN
	short* ifdata,					// if ������ (5ms)
	int* CA_code_GPS_1023,
	short* dopp_freq,
	hipfftDoubleComplex* replica_carr_vec,
	hipfftDoubleComplex* replica_code_vec,
	hipfftDoubleComplex* fft_out_carr_vec,
	hipfftDoubleComplex* fft_out_code_vec,
	hipfftDoubleComplex* freq_conj_carr_code,
	hipfftDoubleComplex* dump_IFFT,
	double* Z_IFFT,
	double* Cpu_Z_IFFT,
	int* ACQ_IF_DUMP,
	int* ACQ_MODE_V2,
	int* ACTIVECHANNEL,
	int* Cpu_code_phase
)
{
	int c = 0;
	int sample_size_5ms = (int)(SAMPLING_FREQ * 5 / 1000.0);

	// Use 5ms grid configuration
	dim3 grids(blocksPerGrid_5ms, doppler_bin_size * 32, 1);

	hipfftHandle p_cufft1, p_cufft2, p_cuifft;

	// Generate 5ms replicas
	f_Replica_Gen_5ms<<<grids, threadsPerBlock>>>(replica_carr_vec, replica_code_vec, ifdata, dopp_freq, CA_code_GPS_1023, PRN);

	int batch = doppler_bin_size * 32;

	// FFT Plan for 5ms data
	hipfftPlan1d(&p_cufft1, sample_size_5ms, HIPFFT_Z2Z, batch);
	hipfftPlan1d(&p_cufft2, sample_size_5ms, HIPFFT_Z2Z, batch);
	hipfftPlan1d(&p_cuifft, sample_size_5ms, HIPFFT_Z2Z, batch);

	// FFT execution
	hipfftExecZ2Z(p_cufft1, replica_carr_vec, fft_out_carr_vec, HIPFFT_FORWARD);
	hipfftExecZ2Z(p_cufft2, replica_code_vec, fft_out_code_vec, HIPFFT_FORWARD);

	hipDeviceSynchronize();

	// Frequency domain conjugate multiplication for 5ms
	f_Freq_Conjugate_5ms<<<grids, threadsPerBlock>>>(freq_conj_carr_code, fft_out_carr_vec, fft_out_code_vec);

	// IFFT
	hipfftExecZ2Z(p_cuifft, freq_conj_carr_code, dump_IFFT, HIPFFT_BACKWARD);

	hipDeviceSynchronize();

	// Calculate absolute values for 5ms
	f_Kernel_Abs_5ms<<<grids, threadsPerBlock>>>(dump_IFFT, Z_IFFT);

	// Copy results back to host (5ms size)
	hipMemcpy(Cpu_Z_IFFT, Z_IFFT, sizeof(double) * sample_size_5ms * doppler_bin_size * 32, hipMemcpyDeviceToHost);

	while ((*ACQ_IF_DUMP) == 1 && (*ACQ_MODE_V2) == 1)
	{
		// Use 1ms peak detector
		f_Peak_Detector(CH[c], Cpu_Z_IFFT);

		if (CH[c].detect == true)
		{
			(*ACTIVECHANNEL)++;
			Cpu_code_phase[c] = CH[c].codephase;

			if (start_mode == false)
			{
				if (CH[c].PRN < 32)
				{
					CH[c + 1].PRN = CH[c].PRN + 1;
					c++;
				}
				else
				{
					(*ACQ_MODE_V2) = 0;
				}
			}
			else
			{
				c++;

				if (c == (*ACTIVECHANNEL))
				{
					(*ACQ_MODE_V2) = 0;
				}
			}
		}
		else
		{
			if (CH[c].PRN < 32)
			{
				CH[c].PRN++;
			}
			else
			{
				(*ACQ_MODE_V2) = 0;
			}
		}

		(*ACQ_IF_DUMP) = 0;	// reset flag
	}

	hipfftDestroy(p_cufft1);
	hipfftDestroy(p_cufft2);
	hipfftDestroy(p_cuifft);
}


__global__ void kernel_Correlation_First(short* PRN, short* code_index, short* carr_temp, int* s_c, short* ifdata, float* dump, int* cuda_CA_code_GPS_2046, int* trk_CH_2046)
{
	__shared__ int cache_dump_I_E[threadsPerBlock];		// ���� �޸� �Ҵ� -> �� �����尡 �ջ��� ����� �����ϴµ� ���
	__shared__ int cache_dump_I_P[threadsPerBlock];
	__shared__ int cache_dump_I_L[threadsPerBlock];
	__shared__ int cache_dump_Q_E[threadsPerBlock];
	__shared__ int cache_dump_Q_P[threadsPerBlock];
	__shared__ int cache_dump_Q_L[threadsPerBlock];

	int thread_num = threadIdx.x + (blockIdx.x * blockDim.x);
	int CH = blockIdx.y;

	short IF_DATA;
	bool Dump_Sync_Check = false;
	int thread_code_index = threadIdx.x + (blockIdx.x * blockDim.x) + ((int)(CorrSizeLocal * 2) * CH);

	int temp_I_E, temp_I_P, temp_I_L, temp_Q_E, temp_Q_P, temp_Q_L;
	int code_e, code_p, code_l;
	int carr_I, carr_Q;

	temp_I_E = temp_I_P = temp_I_L = temp_Q_E = temp_Q_P = temp_Q_L = carr_I = carr_Q = 0.0;
	code_e = code_p = code_l = 0;


	if (thread_num > trk_CH_2046[CH * 2])
		Dump_Sync_Check = true;


	IF_DATA = ifdata[thread_num + (int)(CorrSizeLocal * 2) * CH] >> 8;

	if (Dump_Sync_Check == false)
	{
		code_e = cuda_CA_code_GPS_2046[((PRN[CH]) - 1) * 2046 + ((code_index[thread_code_index] + 1 + s_c[CH]) + 2046) % 2046];
		code_p = cuda_CA_code_GPS_2046[((PRN[CH]) - 1) * 2046 + ((code_index[thread_code_index] + s_c[CH]) + 2046) % 2046];
		code_l = cuda_CA_code_GPS_2046[((PRN[CH]) - 1) * 2046 + ((code_index[thread_code_index] - 1 + s_c[CH]) + 2046) % 2046];
	}
	else
	{
		code_e = cuda_CA_code_GPS_2046[((PRN[CH]) - 1) * 2046 + ((code_index[thread_code_index] + 1) + 2046) % 2046];
		code_p = cuda_CA_code_GPS_2046[((PRN[CH]) - 1) * 2046 + ((code_index[thread_code_index]) + 2046) % 2046];
		code_l = cuda_CA_code_GPS_2046[((PRN[CH]) - 1) * 2046 + ((code_index[thread_code_index] - 1) + 2046) % 2046];
	}

	switch (carr_temp[thread_code_index])
	{
	case 0:
		carr_I = 1;
		carr_Q = 2;
		break;
	case 1:
		carr_I = 2;
		carr_Q = 1;
		break;
	case 2:
		carr_I = 2;
		carr_Q = -1;
		break;
	case 3:
		carr_I = 1;
		carr_Q = -2;
		break;
	case 4:
		carr_I = -1;
		carr_Q = -2;
		break;
	case 5:
		carr_I = -2;
		carr_Q = -1;
		break;
	case 6:
		carr_I = -2;
		carr_Q = 1;
		break;
	default:
		carr_I = -1;
		carr_Q = 2;
	}



	__syncthreads();

	if (thread_num <= trk_CH_2046[CH * 2 + 1])
	{
		temp_I_E = IF_DATA * carr_I * code_e;
		temp_I_P = IF_DATA * carr_I * code_p;
		temp_I_L = IF_DATA * carr_I * code_l;
		temp_Q_E = IF_DATA * carr_Q * code_e;
		temp_Q_P = IF_DATA * carr_Q * code_p;
		temp_Q_L = IF_DATA * carr_Q * code_l;
	}
	else
	{
		temp_I_E = 0;
		temp_I_P = 0;
		temp_I_L = 0;
		temp_Q_E = 0;
		temp_Q_P = 0;
		temp_Q_L = 0;
	}

	__syncthreads();

	cache_dump_I_E[threadIdx.x] = 0;
	cache_dump_I_P[threadIdx.x] = 0;
	cache_dump_I_L[threadIdx.x] = 0;
	cache_dump_Q_E[threadIdx.x] = 0;
	cache_dump_Q_P[threadIdx.x] = 0;
	cache_dump_Q_L[threadIdx.x] = 0;

	__syncthreads();

	cache_dump_I_E[threadIdx.x] = temp_I_E;
	cache_dump_I_P[threadIdx.x] = temp_I_P;
	cache_dump_I_L[threadIdx.x] = temp_I_L;
	cache_dump_Q_E[threadIdx.x] = temp_Q_E;
	cache_dump_Q_P[threadIdx.x] = temp_Q_P;
	cache_dump_Q_L[threadIdx.x] = temp_Q_L;

	__syncthreads();


	int i = blockDim.x / 2;
	while (i != 0)
	{
		if (threadIdx.x < i)
		{
			cache_dump_I_E[threadIdx.x] += cache_dump_I_E[threadIdx.x + i];
			cache_dump_I_P[threadIdx.x] += cache_dump_I_P[threadIdx.x + i];
			cache_dump_I_L[threadIdx.x] += cache_dump_I_L[threadIdx.x + i];
			cache_dump_Q_E[threadIdx.x] += cache_dump_Q_E[threadIdx.x + i];
			cache_dump_Q_P[threadIdx.x] += cache_dump_Q_P[threadIdx.x + i];
			cache_dump_Q_L[threadIdx.x] += cache_dump_Q_L[threadIdx.x + i];

		}
		__syncthreads();


		i /= 2;
	}


	if (threadIdx.x == 0)
	{
		dump[(CH * gridDim.x * 6) + blockIdx.x] = cache_dump_I_E[0];
		dump[(CH * gridDim.x * 6) + (gridDim.x * 1) + blockIdx.x] = cache_dump_I_P[0];
		dump[(CH * gridDim.x * 6) + (gridDim.x * 2) + blockIdx.x] = cache_dump_I_L[0];
		dump[(CH * gridDim.x * 6) + (gridDim.x * 3) + blockIdx.x] = cache_dump_Q_E[0];
		dump[(CH * gridDim.x * 6) + (gridDim.x * 4) + blockIdx.x] = cache_dump_Q_P[0];
		dump[(CH * gridDim.x * 6) + (gridDim.x * 5) + blockIdx.x] = cache_dump_Q_L[0];

	}

}

__global__ void kernel_Correlation2(short* PRN, short* code_index, short* carr_temp, short* ifdata, float* dump, int* cuda_CA_code_GPS_2046, int* trk_CH_2046, int* trk_count)
{
	__shared__ int cache_dump_I_E[threadsPerBlock];		// ���� �޸� �Ҵ� -> �� �����尡 �ջ��� ����� �����ϴµ� ���
	__shared__ int cache_dump_I_P[threadsPerBlock];
	__shared__ int cache_dump_I_L[threadsPerBlock];
	__shared__ int cache_dump_Q_E[threadsPerBlock];
	__shared__ int cache_dump_Q_P[threadsPerBlock];
	__shared__ int cache_dump_Q_L[threadsPerBlock];
	
	int thread_num = threadIdx.x + (blockIdx.x * blockDim.x);
	int CH = blockIdx.y;
	int shift_thread_num = thread_num + trk_CH_2046[2 * CH] + 1;
	

	//if (thread_num > (trk_CH_2046[2 * CH + 1] - trk_CH_2046[2 * CH]))	return;

	short IF_DATA;
	//bool Dump_Sync_Check = false;
	int thread_code_index = thread_num + ((int)CorrSizeLocal * 2 * CH);
	//int thread_code_index = shift_thread_num + ((int)CorrSizeLocal * 2 * CH);
	
	int test_I_E, test_I_P, test_I_L, test_Q_E, test_Q_P, test_Q_L;

	int temp_I_E, temp_I_P, temp_I_L, temp_Q_E, temp_Q_P, temp_Q_L;
	int code_e, code_p, code_l;
	int carr_I, carr_Q;
	//short IF_DATA = 0;

	temp_I_E = temp_I_P = temp_I_L = temp_Q_E = temp_Q_P = temp_Q_L = carr_I = carr_Q = 0.0;
	code_e = code_p = code_l = 0;


	//if (thread_num > trk_CH_2046[CH * 2])
	//	Dump_Sync_Check == true;

	IF_DATA = ifdata[thread_num + (int)(CorrSizeLocal * 2) * CH] >> 8;
	//IF_DATA = ifdata[shift_thread_num] >> 8;


	//if (Dump_Sync_Check == false)
	//{
	//	code_e = cuda_CA_code_GPS_2046[((PRN[CH]) - 1) * 2046 + ((code_index[thread_code_index] + 1 + s_c[CH] * 2) + 2046) % 2046];
	//	code_p = cuda_CA_code_GPS_2046[((PRN[CH]) - 1) * 2046 + ((code_index[thread_code_index] + s_c[CH] * 2) + 2046) % 2046];
	//	code_l = cuda_CA_code_GPS_2046[((PRN[CH]) - 1) * 2046 + ((code_index[thread_code_index] - 1 + s_c[CH] * 2) + 2046) % 2046];
	//}
	
	
	code_e = cuda_CA_code_GPS_2046[((PRN[CH]) - 1) * 2046 + ((code_index[thread_code_index] + 1) + 2046) % 2046];
	code_p = cuda_CA_code_GPS_2046[((PRN[CH]) - 1) * 2046 + ((code_index[thread_code_index]) + 2046) % 2046];
	code_l = cuda_CA_code_GPS_2046[((PRN[CH]) - 1) * 2046 + ((code_index[thread_code_index] - 1) + 2046) % 2046];
	

	switch (carr_temp[thread_code_index])
	{
	case 0:
		carr_I = 1;
		carr_Q = 2;
		break;
	case 1:
		carr_I = 2;
		carr_Q = 1;
		break;
	case 2:
		carr_I = 2;
		carr_Q = -1;
		break;
	case 3:
		carr_I = 1;
		carr_Q = -2;
		break;
	case 4:
		carr_I = -1;
		carr_Q = -2;
		break;
	case 5:
		carr_I = -2;
		carr_Q = -1;
		break;
	case 6:
		carr_I = -2;
		carr_Q = 1;
		break;
	default:
		carr_I = -1;
		carr_Q = 2;
	}

	__syncthreads();

	if (thread_num <= trk_CH_2046[CH * 2 + 1])
	//if (shift_thread_num <= trk_CH_2046[CH * 2 + 1])
	{
		temp_I_E = IF_DATA * carr_I * code_e;
		temp_I_P = IF_DATA * carr_I * code_p;
		temp_I_L = IF_DATA * carr_I * code_l;
		temp_Q_E = IF_DATA * carr_Q * code_e;
		temp_Q_P = IF_DATA * carr_Q * code_p;
		temp_Q_L = IF_DATA * carr_Q * code_l;
	}
	else
	{
		temp_I_E = 0;
		temp_I_P = 0;
		temp_I_L = 0;
		temp_Q_E = 0;
		temp_Q_P = 0;
		temp_Q_L = 0;
	}

	__syncthreads();

	cache_dump_I_E[threadIdx.x] = 0;
	cache_dump_I_P[threadIdx.x] = 0;
	cache_dump_I_L[threadIdx.x] = 0;
	cache_dump_Q_E[threadIdx.x] = 0;
	cache_dump_Q_P[threadIdx.x] = 0;
	cache_dump_Q_L[threadIdx.x] = 0;

	__syncthreads();

	
	cache_dump_I_E[threadIdx.x] = temp_I_E;
	cache_dump_I_P[threadIdx.x] = temp_I_P;
	cache_dump_I_L[threadIdx.x] = temp_I_L;
	cache_dump_Q_E[threadIdx.x] = temp_Q_E;
	cache_dump_Q_P[threadIdx.x] = temp_Q_P;
	cache_dump_Q_L[threadIdx.x] = temp_Q_L;

	__syncthreads();

	
	int i = blockDim.x / 2;
	while (i != 0)
	{
		if (threadIdx.x < i)
		{
			cache_dump_I_E[threadIdx.x] += cache_dump_I_E[threadIdx.x + i];
			cache_dump_I_P[threadIdx.x] += cache_dump_I_P[threadIdx.x + i];
			cache_dump_I_L[threadIdx.x] += cache_dump_I_L[threadIdx.x + i];
			cache_dump_Q_E[threadIdx.x] += cache_dump_Q_E[threadIdx.x + i];
			cache_dump_Q_P[threadIdx.x] += cache_dump_Q_P[threadIdx.x + i];
			cache_dump_Q_L[threadIdx.x] += cache_dump_Q_L[threadIdx.x + i];
		}
		__syncthreads();
		i /= 2;
	}

	if (threadIdx.x == 0)
	{
		dump[(blockIdx.y * gridDim.x * 6) + blockIdx.x] = cache_dump_I_E[0];
		dump[(blockIdx.y * gridDim.x * 6) + (gridDim.x * 1) + blockIdx.x] = cache_dump_I_P[0];
		dump[(blockIdx.y * gridDim.x * 6) + (gridDim.x * 2) + blockIdx.x] = cache_dump_I_L[0];
		dump[(blockIdx.y * gridDim.x * 6) + (gridDim.x * 3) + blockIdx.x] = cache_dump_Q_E[0];
		dump[(blockIdx.y * gridDim.x * 6) + (gridDim.x * 4) + blockIdx.x] = cache_dump_Q_P[0];
		dump[(blockIdx.y * gridDim.x * 6) + (gridDim.x * 5) + blockIdx.x] = cache_dump_Q_L[0];

	}

}

void f_Correlation
(
	short* PRN,						// ���� PRN
	short* code_index,
	short* carr_temp,
	int* s_c,
	short* ifdata,					// if ������
	float* dump,
	int* cuda_CA_code_GPS_2046,
	int ACTIVECHANNEL,
	int* trk_CH_2046,
	int TRK_first,
	int* trk_count
)
{

	if(TRK_first == 1)
	{	
		dim3 grids_trk(First_trk_blocksPerGrid, ACTIVECHANNEL);
		kernel_Correlation_First << <grids_trk, threadsPerBlock >> > (PRN, code_index, carr_temp, s_c, ifdata, dump, cuda_CA_code_GPS_2046, trk_CH_2046);

	}
	else
	{
		dim3 grids_trk(trk_blocksPerGrid, ACTIVECHANNEL);
		kernel_Correlation2 << <grids_trk, threadsPerBlock >> > (PRN, code_index, carr_temp, ifdata, dump, cuda_CA_code_GPS_2046, trk_CH_2046, trk_count);
	}

}

void f_constant_setup(int setup_f_s, int setup_f_if_GPS, int setup_f_ca_GPS, unsigned int sample_size_1ms)
{
	hipMemcpyToSymbol(HIP_SYMBOL(Gpu_const_f_s), &setup_f_s, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(Gpu_const_f_if_GPS), &setup_f_if_GPS, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(Gpu_const_f_ca_GPS), &setup_f_ca_GPS, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(Gpu_const_sample_size_1ms), &sample_size_1ms, sizeof(unsigned long));

}